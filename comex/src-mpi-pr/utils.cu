#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/* avoid name mangling by the CUDA compiler */
extern "C" {

/* return the number of GPUs visible from this processor*/
int numDevices()
{
  int ngpus;
  hipError_t err;
  err = hipGetDeviceCount(&ngpus);
  /*cuDeviceGetCount(&ngpus); */
  if (err != hipSuccess) {
    printf("Error encountered by cudaGetDeviceCount\n");
  }
  return ngpus;
}

/* set the GPU device for this processor
 * id: id of device
 */
void setDevice(int id)
{
  hipSetDevice(id);
}

/* allocate a unified memory buffer
 * buf: pointer to buffer
 * size: size of allocation in bytes
 */
void mallocDevice(void **buf, size_t size)
{
  hipPointerAttribute_t attr;
  hipError_t err;
  hipMalloc(buf, (int)size);
  err = hipPointerGetAttributes(&attr, *buf);
}

/* free unified memory
 * buf: pointer to memory allocation
 */
void freeDevice(void *buf)
{
  hipFree(buf);
}

/* copy data from host buffer to unified memory
 * hostptr: pointer to allocation on host
 * devptr: pointer to allocation on device
 * bytes: number of bytes to copy
 */
void copyToDevice(void *hostptr, void *devptr, int bytes)
{
  hipMemcpy(devptr, hostptr, bytes, hipMemcpyHostToDevice); 
}

/* copy data from unified memory to host buffer
 * hostptr: pointer to allocation on host
 * devptr: pointer to allocation on device
 * bytes: number of bytes to copy
 */
void copyToHost(void *hostptr, void *devptr, int bytes)
{
  hipMemcpy(hostptr, devptr, bytes, hipMemcpyDeviceToHost); 
}

/* copy data between devices using unified memory
 * srcptr: source pointer
 * dstptr: destination pointer
 * bytes: number of bytes to copy
 */
void copyDevToDev(void *srcptr, void *dstptr, int bytes)
{
  hipMemcpy(dstptr, srcptr, bytes, hipMemcpyDeviceToDevice); 
}

/**
 * set values on the device to a specific value
 * ptr: pointer to device memory that needs to be set
 * val: integer representation of the value of each byte
 * size: number of bytes that should be set
 */
void deviceMemset(void *ptr, int val, size_t bytes)
{
  hipMemset(ptr, val, bytes);
}

/* is pointer located on host?
 * return 1 is data is only located on host, 0 otherwise
 * ptr: pointer to data
 */
int isHostPointer(void *ptr)
{
  hipPointerAttribute_t attr;
  hipError_t  err = hipPointerGetAttributes(&attr, ptr);
  /* Assume that if Cuda doesn't know anything about the pointer, it is on the
   * host */
  if (err != hipSuccess) return 1;
  if (attr.devicePointer == NULL) {
    return  1;
  }
  return 0;
}

__global__ void iaxpy_kernel(int *dst, const int *src, int scale)
{
  int i = threadIdx.x;

  dst[i] = dst[i] + scale*src[i];
}

void deviceIaxpy(int *dst, const int *src, const int *scale, int n)
{
  iaxpy_kernel<<<1,n>>>(dst, src, *scale);
}

};
