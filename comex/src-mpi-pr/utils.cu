#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/* avoid name mangling by the CUDA compiler */
extern "C" {

/* return the number of GPUs visible from this processor */
int numDevices()
{
  int ngpus;
  hipGetDeviceCount(&ngpus);
  return ngpus;
}

/* set the GPU device for this processor */
void setDevice(int id)
{
  hipSetDevice(id);
}

/* allocate a unified memory buffer */
void mallocDevice(void **buf, size_t size)
{
  hipMallocManaged(buf, (int)size);
}

/* free unified memory */
void freeDevice(void *buf)
{
  hipFree(buf);
}

/* copy data from host buffer to unified memory */
void copyToDevice(void *hostptr, void *devptr, int bytes)
{
  hipMemcpy(devptr, hostptr, bytes, hipMemcpyHostToDevice); 
}

/* copy data from unified memory to host buffer */
void copyToHost(void *hostptr, void *devptr, int bytes)
{
  hipMemcpy(hostptr, devptr, bytes, hipMemcpyDeviceToHost); 
}

};
